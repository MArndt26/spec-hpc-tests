#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "../../shared.h"
#include "pack_halos.cuknl"

typedef void (*pack_kernel_f)( 
        const int, const int, const int, const int, 
        double*, double*, const int);

// Either packs or unpacks data from/to buffers.
void pack_or_unpack(
        Chunk* chunk, Settings* settings, int depth, int face, 
        bool pack, double* field, double* buffer)
{
    pack_kernel_f kernel = NULL;

    const int x_inner = x - 2*halo_depth;
    const int y_inner = y - 2*halo_depth;
    const int z_inner = z - 2*halo_depth;

    int buffer_length = 0;

    switch(face)
    {
        case CHUNK_LEFT:
            kernel = pack ? pack_left : unpack_left;
            buffer_length = y_inner*z_inner*depth;
            break;
        case CHUNK_RIGHT:
            kernel = pack ? pack_right : unpack_right;
            buffer_length = y_inner*z_inner*depth;
            break;
        case CHUNK_TOP:
            kernel = pack ? pack_top : unpack_top;
            buffer_length = x_inner*z_inner*depth;
            break;
        case CHUNK_BOTTOM:
            kernel = pack ? pack_bottom : unpack_bottom;
            buffer_length = x_inner*z_inner*depth;
            break;
        case CHUNK_FRONT:
            kernel = pack ? pack_front : unpack_front;
            buffer_length = x_inner*y_inner*depth;
            break;
        case CHUNK_BACK:
            kernel = pack ? pack_back : unpack_back;
            buffer_length = x_inner*y_inner*depth;
            break;
        default:
            die(__LINE__, __FILE__, "Incorrect face provided: %d.\n", face);
    }

    if(!pack)
    {
        hipMemcpy(
                d_comm_buffer, buffer, buffer_length*sizeof(double), 
                hipMemcpyHostToDevice);
        check_errors(__LINE__, __FILE__);
    }

    int num_blocks = ceil(buffer_length / (double)BLOCK_SIZE);
    kernel<<<num_blocks, BLOCK_SIZE>>>(
            x, y, z, depth, halo_depth, field, chunk->d_comm_buffer);

    if(pack)
    {
        hipMemcpy(
                d_comm_buffer, buffer, buffer_length*sizeof(double),
                hipMemcpyDeviceToHost);
        check_errors(__LINE__, __FILE__);
    }
}

