#include "hip/hip_runtime.h"
#include "../../kernel_interface.h"
#include "c_kernels.h"
#include "cudaknl_shared.h"

#define PRE_KERNEL(pad) \
    const int x_inner = chunk->x - 2*pad; \
    const int y_inner = chunk->y - 2*pad; \
    const int z_inner = chunk->z - 2*pad; \
    const int num_threads = x_inner*y_inner*z_inner; \
    const int num_blocks = ceil((double)num_threads/BLOCK_SIZE); \
    START_PROFILING(settings->kernel_profile);

#define POST_KERNEL(kernel_name) \
    STOP_PROFILING(settings->kernel_profile, kernel_name); \
    check_errors(__LINE__, __FILE__);

void run_set_chunk_data(Chunk* chunk, Settings* settings)
{
    int num_threads = 1 + max(chunk->x, max(chunk->y, chunk->z));
    int num_blocks = ceil((double)num_threads / (double)BLOCK_SIZE);

    START_PROFILING(settings->kernel_profile);

    double x_min = settings->grid_x_min + settings->dx*(double)chunk->left;
    double y_min = settings->grid_y_min + settings->dy*(double)chunk->bottom;
    double z_min = settings->grid_z_min + settings->dz*(double)chunk->back;

    set_chunk_data_vertices<<<num_blocks, BLOCK_SIZE>>>(
            chunk->x, chunk->y, chunk->z, settings->dx,
            settings->dy, settings->dz, x_min, y_min, z_min, 
            chunk->vertex_x, chunk->vertex_y, chunk->vertex_z);

    num_threads = (chunk->x+1)*(chunk->y+1)*(chunk->z+1);
    num_blocks = ceil((double)num_threads / (double)BLOCK_SIZE);

    set_chunk_data<<<num_blocks, BLOCK_SIZE>>>(
            chunk->x, chunk->y, chunk->z, settings->dx,
            settings->dy, settings->dz, chunk->cell_x, chunk->cell_y,
            chunk->cell_z, chunk->vertex_x, chunk->vertex_y, chunk->vertex_z,
            chunk->volume, chunk->x_area, chunk->y_area, chunk->z_area);

    STOP_PROFILING(settings->kernel_profile, "set_chunk_data");
}

void run_set_chunk_state(Chunk* chunk, Settings* settings, State* states)
{
    PRE_KERNEL(0);

    set_chunk_initial_state<<<num_blocks, BLOCK_SIZE>>>(
            chunk->x, chunk->y, chunk->z, states[0].energy, 
            states[0].density, chunk->energy0, chunk->density);

    set_chunk_state<<<num_blocks, BLOCK_SIZE>>>(
            chunk->x, chunk->y, chunk->z, chunk->vertex_x,
            chunk->vertex_y, chunk->vertex_z, chunk->cell_x, chunk->cell_y,
            chunk->cell_z, chunk->density, chunk->energy0, chunk->vec_u,
            state[ii]);

    POST_KERNEL("set_chunk_state");
}

void run_kernel_initialise(Chunk* chunk, Settings* settings)
{
    kernel_initialise(settings, chunk->x, chunk->y, chunk->z, &(chunk->density0), 
            &(chunk->density), &(chunk->energy0), &(chunk->energy), 
            &(chunk->vec_u), &(chunk->vec_u0), &(chunk->vec_p), &(chunk->vec_r), 
            &(chunk->vec_mi), &(chunk->vec_w), &(chunk->vec_kx), &(chunk->vec_ky), 
            &(chunk->vec_kz), &(chunk->vec_sd), &(chunk->vec_z), &(chunk->volume), 
            &(chunk->x_area), &(chunk->y_area), &(chunk->z_area), 
            &(chunk->cell_x), &(chunk->cell_y), &(chunk->cell_z), 
            &(chunk->cell_dx), &(chunk->cell_dy), &(chunk->cell_dz), 
            &(chunk->vertex_dx), &(chunk->vertex_dy), &(chunk->vertex_dz), 
            &(chunk->vertex_x), &(chunk->vertex_y), &(chunk->vertex_z), 
            &(chunk->cg_alphas), &(chunk->cg_betas), &(chunk->cheby_alphas),
            &(chunk->cheby_betas));
}

// Solver-wide kernels
void run_local_halos(
        Chunk* chunk, Settings* settings, int depth)
{
    START_PROFILING(settings->kernel_profile);
    local_halos(chunk->x, chunk->y, chunk->z, depth, settings->halo_depth, 
            chunk->neighbours, settings->fields_to_exchange, chunk->density,
            chunk->energy0, chunk->energy, chunk->vec_u, chunk->vec_p, 
            chunk->vec_sd);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_pack_or_unpack(
        Chunk* chunk, Settings* settings, int depth,
        int face, bool pack, double* field, double* buffer)
{
    START_PROFILING(settings->kernel_profile);
    pack_or_unpack(chunk->x, chunk->y, chunk->z, depth, 
            settings->halo_depth, face, pack, field, buffer);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_store_energy(Chunk* chunk, Settings* settings)
{
    PRE_KERNEL(0);

    store_energy<<<num_blocks, BLOCK_SIZE>>>(
            x_inner, y_inner, z_inner, chunk->energy0, chunk->energy);

    POST_KERNEL("store_energy");
}

void run_field_summary(
        Chunk* chunk, Settings* settings, 
        double* vol, double* mass, double* ie, double* temp)
{
    PRE_KERNEL(settings->halo_depth*2);

    field_summary<<<num_blocks, BLOCK_SIZE>>>(
            x_inner, y_inner, z_inner, settings->halo_depth, 
            chunk->volume, chunk->density, chunk->energy0, 
            chunk->vec_u, vol, mass, ie, temp);

    POST_KERNEL("field_summary");

    sum_reduce_buffer(
}

// CG solver kernels
void run_cg_solver_init(
        Chunk* chunk, Settings* settings, 
        double rx, double ry, double rz, double* rro)
{
    START_PROFILING(settings->kernel_profile);
    cg_solver_init(chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, settings->coefficient, rx, ry, rz, 
            rro, chunk->density, chunk->energy, chunk->vec_u, 
            chunk->vec_p, chunk->vec_r, chunk->vec_w, 
            chunk->vec_kx, chunk->vec_ky, chunk->vec_kz);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_cg_solver_calc_w(Chunk* chunk, Settings* settings, double* pw)
{
    START_PROFILING(settings->kernel_profile);
    cg_solver_calc_w(chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, pw, chunk->vec_p, 
            chunk->vec_w, chunk->vec_kx,
            chunk->vec_ky, chunk->vec_kz);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_cg_solver_calc_ur(
        Chunk* chunk, Settings* settings, double alpha, double* rrn)
{
    START_PROFILING(settings->kernel_profile);
    cg_solver_calc_ur(chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, alpha, rrn, chunk->vec_u, 
            chunk->vec_p, chunk->vec_r, chunk->vec_w);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_cg_solver_calc_p(Chunk* chunk, Settings* settings, double beta)
{
    START_PROFILING(settings->kernel_profile);
    cg_solver_calc_p(chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, beta, chunk->vec_p, 
            chunk->vec_r);
    STOP_PROFILING(settings->kernel_profile, __func__);
}


// Chebyshev solver kernels
void run_cheby_solver_init(Chunk* chunk, Settings* settings)
{
    START_PROFILING(settings->kernel_profile);
    cheby_solver_init(
            chunk->x, chunk->y, chunk->z, settings->halo_depth, 
            chunk->theta, chunk->vec_u, chunk->vec_u0, 
            chunk->vec_p, chunk->vec_r, chunk->vec_w, 
            chunk->vec_kx, chunk->vec_ky, chunk->vec_kz);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_cheby_solver_iterate(
        Chunk* chunk, Settings* settings, double alpha, double beta)
{
    START_PROFILING(settings->kernel_profile);
    cheby_solver_iterate(
            chunk->x, chunk->y, chunk->z, settings->halo_depth, alpha, beta, 
            chunk->vec_u, chunk->vec_u0, chunk->vec_p, chunk->vec_r, chunk->vec_w, 
            chunk->vec_kx, chunk->vec_ky, chunk->vec_kz); 
    STOP_PROFILING(settings->kernel_profile, __func__);
}


// Jacobi solver kernels
void run_jacobi_solver_init(
        Chunk* chunk, Settings* settings, double rx, double ry, double rz)
{
    START_PROFILING(settings->kernel_profile);
    jacobi_solver_init(chunk->x, chunk->y, chunk->z,
            settings->halo_depth, settings->coefficient, rx, ry, rz,
            chunk->density, chunk->energy, chunk->vec_u0,
            chunk->vec_u, chunk->vec_kx, chunk->vec_ky,
            chunk->vec_kz);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_jacobi_solver_iterate(
        Chunk* chunk, Settings* settings, double* error)
{
    START_PROFILING(settings->kernel_profile);
    jacobi_solver_iterate(
            chunk->x, chunk->y, chunk->z,
            settings->halo_depth, error, chunk->vec_kx, 
            chunk->vec_ky, chunk->vec_kz, chunk->vec_u0, 
            chunk->vec_u, chunk->vec_r);
    STOP_PROFILING(settings->kernel_profile, __func__);
}


// PPCG solver kernels
void run_ppcg_init(Chunk* chunk, Settings* settings)
{
    START_PROFILING(settings->kernel_profile);
    ppcg_init(chunk->x, chunk->y, chunk->z,
            settings->halo_depth, chunk->theta, chunk->vec_r,
            chunk->vec_sd);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_ppcg_inner_iteration(
        Chunk* chunk, Settings* settings, double alpha, double beta)
{
    START_PROFILING(settings->kernel_profile);
    ppcg_inner_iteration(
            chunk->x, chunk->y, chunk->z, settings->halo_depth, alpha, beta, 
            chunk->vec_u, chunk->vec_r, chunk->vec_kx, chunk->vec_ky,
            chunk->vec_kz, chunk->vec_sd);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

// Shared solver kernels
void run_solver_copy_u(Chunk* chunk, Settings* settings)
{
    START_PROFILING(settings->kernel_profile);
    solver_copy_u(chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, chunk->vec_u0, chunk->vec_u);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_calculate_residual(Chunk* chunk, Settings* settings)
{
    START_PROFILING(settings->kernel_profile);
    calculate_residual(chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, chunk->vec_u, chunk->vec_u0, 
            chunk->vec_r, chunk->vec_kx, chunk->vec_ky, 
            chunk->vec_kz);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_calculate_2norm(
        Chunk* chunk, Settings* settings, double* buffer, double* norm)
{
    START_PROFILING(settings->kernel_profile);
    calculate_2norm(
            chunk->x, chunk->y, chunk->z, 
            settings->halo_depth, buffer, norm);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

void run_solver_finalise(Chunk* chunk, Settings* settings)
{
    START_PROFILING(settings->kernel_profile);
    solver_finalise(chunk->x, chunk->y, chunk->z, settings->halo_depth,
            chunk->energy, chunk->density, chunk->vec_u);
    STOP_PROFILING(settings->kernel_profile, __func__);
}

