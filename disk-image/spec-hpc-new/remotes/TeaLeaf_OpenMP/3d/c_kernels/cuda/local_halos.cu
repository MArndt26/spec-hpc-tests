#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "../../shared.h"
#include "local_halos.cuknl"

/*
 * 		LOCAL HALOS KERNEL
 */	

// The kernel for updating halos locally
void local_halos(
        const int x,
        const int y,
        const int z,
        const int depth,
        const int halo_depth,
        const int* chunk_neighbours,
        const bool* fields_to_exchange,
        double* density,
        double* energy0,
        double* energy,
        double* vec_u,
        double* vec_p,
        double* vec_sd)
{
#define LAUNCH_UPDATE(index, buffer)\
    if(fields_to_exchange[index])\
    {\
        update_face(x, y, z, halo_depth, chunk_neighbours, depth, buffer);\
    }

    LAUNCH_UPDATE(FIELD_DENSITY, density);
    LAUNCH_UPDATE(FIELD_P, vec_p);
    LAUNCH_UPDATE(FIELD_ENERGY0, energy0);
    LAUNCH_UPDATE(FIELD_ENERGY1, energy);
    LAUNCH_UPDATE(FIELD_U, vec_u);
    LAUNCH_UPDATE(FIELD_SD, vec_sd);
#undef LAUNCH_UPDATE
}

// Updates faces in turn.
void update_face(
        const int x,
        const int y, 
        const int z,
        const int halo_depth,
        const int* chunk_neighbours,
        const int depth,
        double* buffer)
{
    const int x_inner = x - 2*halo_depth;
    const int y_inner = y - 2*halo_depth;
    const int z_inner = z - 2*halo_depth;

#define UPDATE_FACE(face, update_kernel) \
    if(chunk_neighbours[face] == EXTERNAL_FACE) \
    {\
        update_kernel<<<num_blocks, BLOCK_SIZE>>>( \
                x, y, z, halo_depth, depth, buffer); \
    }

    int num_blocks = ceil((x_inner*z_inner*depth) / (double)BLOCK_SIZE);
    UPDATE_FACE(CHUNK_TOP, update_top);
    UPDATE_FACE(CHUNK_BOTTOM, update_bottom);

    num_blocks = ceil((x_inner*y_inner*depth) / (double)BLOCK_SIZE);
    UPDATE_FACE(CHUNK_FRONT, update_front);
    UPDATE_FACE(CHUNK_BACK, update_back);

    num_blocks = ceil((y_inner*z_inner*depth) / (double)BLOCK_SIZE);
    UPDATE_FACE(CHUNK_LEFT, update_left);
    UPDATE_FACE(CHUNK_RIGHT, update_right);
}

